﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void mmKernel(float *c, const float *a, const float *b)
{
    // perform matrix multiply here!
}

int main()
{
    const int n = 5;
    float* A = (float*)malloc(n*n*sizeof(float));
    float* B = (float*)malloc(n * n * sizeof(float));
    float* C = (float*)malloc(n * n * sizeof(float));
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    hipMemcpy(dev_a, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    mmKernel <<<n, n>>> (dev_c, dev_a, dev_b); // n blocks and n threads per block
    hipDeviceSynchronize();

    hipMemcpy(C, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}