#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>

__global__ void gemv_unrolled(float *A, float *x, float *y, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N) {
        float sum = 0.0f;
        for (int col = 0; col < N; col += 4) {
            sum += A[row * N + col] * x[col];
            sum += A[row * N + col + 1] * x[col + 1];
            sum += A[row * N + col + 2] * x[col + 2];
            sum += A[row * N + col + 3] * x[col + 3];
        }
        y[row] = sum;
    }
}

void initialize(float *A, float *x, int N) {
    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < N; ++i) {
        x[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    srand(static_cast<unsigned>(time(0)));
    const int blockSizes[] = {256, 512, 500};
    std::ofstream outFile("gemv_timing_results.csv"); // File stream for writing results to CSV

    // Check if the file was successfully opened.
    if (!outFile.is_open()) {
        std::cerr << "Failed to open the file for writing." << std::endl;
        return -1;
    }

    // Write CSV headers
    outFile << "Matrix Size,Block Size,Time (ms)\n";

    for (int N = 10000; N <= 20000; N += 1000) {
        for (int blockSizeIndex = 0; blockSizeIndex < 3; ++blockSizeIndex) {
            int blockSize = blockSizes[blockSizeIndex];
            float *d_A, *d_x, *d_y;
            hipMalloc(&d_A, N*N*sizeof(float));
            hipMalloc(&d_x, N*sizeof(float));
            hipMalloc(&d_y, N*sizeof(float));

            float *h_A = new float[N*N];
            float *h_x = new float[N];

            initialize(h_A, h_x, N);

            hipMemcpy(d_A, h_A, N*N*sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);

            int numBlocks = (N + blockSize - 1) / blockSize;

            // Start timing
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            gemv_unrolled<<<numBlocks, blockSize>>>(d_A, d_x, d_y, N);

            // Stop timing
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);

            // Write results to the CSV file
            outFile << N << "," << blockSize << "," << elapsedTime << "\n";

            // Cleanup
            hipFree(d_A);
            hipFree(d_x);
            hipFree(d_y);
            delete[] h_A;
            delete[] h_x;

            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }

    outFile.close(); // Close the CSV file
    return 0;
}
